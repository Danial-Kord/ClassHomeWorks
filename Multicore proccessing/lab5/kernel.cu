#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <omp.h>
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, int method);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void addKernelWiththreadSize(int* c, const int* a, const int* b,int n)
{
    int i = threadIdx.x*n;
    int end = i + n;
    for (; i < end; i++) {
        c[i] = a[i] + b[i];
    }
}

__global__ void addKernelWithBlockSize(int* c, const int* a, const int* b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
   c[i] = a[i] + b[i];
}

__global__ void getData(int* block, int* warp, int* thread)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    block[i] = blockIdx.x;
    thread[i] = threadIdx.x;
    warp[i] = threadIdx.x / 32;
}


const int vectorSize = 128;
void fillVector(int* v, size_t n);
void addVector(int* a, int* b, int* c, size_t n);
void printVector(int* v, size_t n);


void serial() {
    
    int a[vectorSize], b[vectorSize], c[vectorSize];

    fillVector(a, vectorSize);
    fillVector(b, vectorSize);



    //serial
    addVector(a, b, c, vectorSize);
    printVector(c, vectorSize);

}

int parallelTaskTest(int method) {

    int a[vectorSize], b[vectorSize], c[vectorSize];

    fillVector(a, vectorSize);
    fillVector(b, vectorSize);


    //   hipEvent_t start, stop;
    //   hipEventRecord(start);
       // Add vectors in parallel.

    //parallel
    hipError_t cudaStatus = addWithCuda(c, a, b, vectorSize,method);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    printVector(c, vectorSize);
    //   hipEventRecord(stop);
    //   float milliseconds = 0;
    //   hipEventElapsedTime(&milliseconds, start, stop);
    //   printf("time: %f",milliseconds);




    return 0;
}

hipError_t calcDataParallel(int* threads, int* warps, int* blocks, unsigned int size, int blockNum, int threadNum)
{
    int* dev_threads = 0;
    int* dev_blocks = 0;
    int* dev_warps = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_threads, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_blocks, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_warps, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    getData << <blockNum, threadNum >> > (dev_blocks, dev_warps, dev_threads);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(blocks, dev_blocks, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(warps, dev_warps, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(threads, dev_threads, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_warps);
    hipFree(dev_blocks);
    hipFree(dev_threads);

    return cudaStatus;
}

void findDataFromGPU() {
    int threads[vectorSize], blocks[vectorSize], warps[vectorSize];
    calcDataParallel(threads,warps,blocks, vectorSize, vectorSize/64,64);
    for (int i = 0; i < vectorSize; i++) {
        printf("calc thread: block: %d, warp: %d, Thread: %d\n", blocks[i], warps[i], threads[i]);
    }
}

int main(){
    double start;
    double end;
    start = omp_get_wtime();


    //serial();
    //parallelTaskTest(2);
    findDataFromGPU();
    end = omp_get_wtime();
    printf("Work took %f seconds\n", end - start);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size,int method=0)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    if (method == 0) {
        // Launch a kernel on the GPU with one thread for each element.
        addKernel << <1, size >> > (dev_c, dev_a, dev_b);
    }
    else if (method == 1) {
        int len = size / 1024;
        addKernelWiththreadSize << <1, 1024 >> > (dev_c, dev_a, dev_b,len);
    }
    else{
        int len = size / 1024;
        addKernelWithBlockSize << <len, 1024 >> > (dev_c, dev_a, dev_b);
    }
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}



// Fills a vector with data
void fillVector(int* v, size_t n) {
    int i;
    for (i = 0; i < n; i++) {
        v[i] = i;
    }
}

// Adds two vectors
void addVector(int* a, int* b, int* c, size_t n) {
#pragma loop(no_vector)
    for (int i = 0; i < n; i++) {
         c[i] = a[i] + b[i];
    }
}

// Prints a vector to the stdout.
void printVector(int* v, size_t n) {
    int i;
    printf("[-] Vector elements: ");
    for (i = 0; i < n; i++) {
        printf("%d, ", v[i]);
    }
    printf("\b\b  \n");
}